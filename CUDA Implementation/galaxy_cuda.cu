#include "hip/hip_runtime.h"
// Melih Kurtaran - Exercise 3 CUDA
// on dione, first load the cuda module
//    module load cuda
//
// compile your program with
//    nvcc -O3 -arch=sm_70 --ptxas-options=-v -o galaxy galaxy_cuda.cu -lm
//
// run your program with
//    srun -p gpu -c 1 --mem=10G ./galaxy_cuda RealGalaxies_100k_arcmin.dat SyntheticGalaxies_100k_arcmin.dat omega.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


float *real_rasc, *real_decl;
float *rand_rasc, *rand_decl;

unsigned long long int *histogramDR, *histogramDD, *histogramRR;
float   pif = acosf(-1.0f); // PI number
long int CPUMemory = 0L;
long int GPUMemory = 0L;

int totaldegrees = 360;
int binsperdegree = 4;

long int NUM_OF_GALAXIES = 100000L;

// put here your GPU kernel(s) to calculate the histograms

//__global__ void  fillHistogram(..) {}
__global__ void fillHistogram(unsigned long long* hist, float* r1_rasc, float* r1_decl, float* r2_rasc, float* r2_decl, long int N)
{
    long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N){
        int j;
        double angle; // angle between galaxies
        for(j=0;j<N;j++)
        {
            angle = 180/acosf(-1.0f)*acosf(sinf(r1_decl[tid])*sinf(r2_decl[j]) + cosf(r1_decl[tid])*cosf(r2_decl[j])*cosf(r1_rasc[tid]-r2_rasc[j]));
            if(isnan(angle)) {angle = 0;}
            atomicAdd(&hist[ (int) floor(angle / 0.25) ],1L);
        }
    }
}

int main(int argc, char *argv[])
{
   int parseargs_readinput(int argc, char *argv[]);
   long int histogramDRsum, histogramDDsum, histogramRRsum;
   double walltime;
   struct timeval _ttime;
   struct timezone _tzone;
   int getDevice(void);
   FILE *outfil;
    
   if ( argc != 4 ) {printf("Usage: a.out real_data random_data output_data\n");return(-1);}

   gettimeofday(&_ttime, &_tzone);
   walltime = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;

    
    // store right ascension and declination for real galaxies here
    // Note: indices run from 0 to 99999 = 100000-1: realrasc[0] -> realrasc[99999]
    // realrasc[100000] is out of bounds for allocated memory!
   real_rasc        = (float *)calloc(NUM_OF_GALAXIES, sizeof(float));
   real_decl        = (float *)calloc(NUM_OF_GALAXIES, sizeof(float));

    // store right ascension and declination for synthetic random galaxies here
   rand_rasc        = (float *)calloc(NUM_OF_GALAXIES, sizeof(float));
   rand_decl        = (float *)calloc(NUM_OF_GALAXIES, sizeof(float));
   CPUMemory += 4L*NUM_OF_GALAXIES*sizeof(float);
    
    if ( parseargs_readinput(argc, argv) != 0 ) {printf("   Program stopped.\n");return(0);}
    printf("   Input data read, now calculating histograms\n");

// For your entertainment: some performance parameters of the GPU you are running your programs on!
   if ( getDevice() != 0 ) return(-1);

    
   histogramDR = (unsigned long long int *)calloc(totaldegrees*binsperdegree+1L,sizeof(unsigned long long int));
   histogramDD = (unsigned long long int *)calloc(totaldegrees*binsperdegree+1L,sizeof(unsigned long long int));
   histogramRR = (unsigned long long int *)calloc(totaldegrees*binsperdegree+1L,sizeof(unsigned long long int));
   CPUMemory += 3L*(totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int);
   
   
   // input data is available in the arrays float real_rasc[], real_decl[], rand_rasc[], rand_decl[];
   // allocate memory on the GPU for input data and histograms
   float* d_real_rasc; hipMalloc(&d_real_rasc, NUM_OF_GALAXIES*sizeof(float));
   float* d_real_decl; hipMalloc(&d_real_decl, NUM_OF_GALAXIES*sizeof(float));
   float* d_rand_rasc; hipMalloc(&d_rand_rasc, NUM_OF_GALAXIES*sizeof(float));
   float* d_rand_decl; hipMalloc(&d_rand_decl, NUM_OF_GALAXIES*sizeof(float));
   GPUMemory += 4L*NUM_OF_GALAXIES*sizeof(float);
    
    
    unsigned long long* d_histogramDR; hipMalloc(&d_histogramDR, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int));
    unsigned long long* d_histogramDD; hipMalloc(&d_histogramDD, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int));
    unsigned long long* d_histogramRR; hipMalloc(&d_histogramRR, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int));
   GPUMemory += 3L*(totaldegrees*binsperdegree+1L)*sizeof(long int);
    
   // and initialize the data on GPU by copying the real and rand data to the GPU
   hipMemcpy(d_real_rasc, real_rasc, NUM_OF_GALAXIES*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_real_decl, real_decl, NUM_OF_GALAXIES*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_rand_rasc, rand_rasc, NUM_OF_GALAXIES*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_rand_decl, rand_decl, NUM_OF_GALAXIES*sizeof(float), hipMemcpyHostToDevice);
   
   // call the GPU kernel(s) that fill the three histograms
   int block_size = 256;
   int grid_size = ((NUM_OF_GALAXIES + block_size) / block_size);
   fillHistogram<<<grid_size,block_size>>>(d_histogramDD, d_real_rasc, d_real_decl, d_real_rasc, d_real_decl, NUM_OF_GALAXIES);
   fillHistogram<<<grid_size,block_size>>>(d_histogramDR, d_real_rasc, d_real_decl, d_rand_rasc, d_rand_decl, NUM_OF_GALAXIES);
   fillHistogram<<<grid_size,block_size>>>(d_histogramRR, d_rand_rasc, d_rand_decl, d_rand_rasc, d_rand_decl, NUM_OF_GALAXIES);
    hipDeviceSynchronize();

   hipMemcpy(histogramDR, d_histogramDR, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int), hipMemcpyDeviceToHost);
   hipMemcpy(histogramDD, d_histogramDD, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int), hipMemcpyDeviceToHost);
   hipMemcpy(histogramRR, d_histogramRR, (totaldegrees*binsperdegree+1L)*sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    

// checking to see if your histograms have the right number of entries
   histogramDRsum = 0L;
   for ( int i = 0; i < binsperdegree*totaldegrees;++i ) histogramDRsum += histogramDR[i];
   printf("   DR histogram sum = %ld\n",histogramDRsum);
   if ( histogramDRsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}

   histogramDDsum = 0L;
   for ( int i = 0; i < binsperdegree*totaldegrees;++i )
        histogramDDsum += histogramDD[i];
   printf("   DD histogram sum = %ld\n",histogramDDsum);
   if ( histogramDDsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}

   histogramRRsum = 0L;
   for ( int i = 0; i < binsperdegree*totaldegrees;++i )
        histogramRRsum += histogramRR[i];
   printf("   RR histogram sum = %ld\n",histogramRRsum);
   if ( histogramRRsum != 10000000000L ) {printf("   Incorrect histogram sum, exiting..\n");return(0);}


   printf("   Omega values:");

   outfil = fopen(argv[3],"w");
   if ( outfil == NULL ) {printf("Cannot open output file %s\n",argv[3]);return(-1);}
   fprintf(outfil,"bin start\tomega\t        hist_DD\t        hist_DR\t        hist_RR\n");
   for ( int i = 0; i < binsperdegree*totaldegrees; ++i )
       {
       if ( histogramRR[i] > 0 )
          {
          double omega =  (histogramDD[i]-2*histogramDR[i]+histogramRR[i])/((double)(histogramRR[i]));

          fprintf(outfil,"%6.3f\t%15lf\t%15ld\t%15ld\t%15ld\n",((float)i)/binsperdegree, omega,
             histogramDD[i], histogramDR[i], histogramRR[i]);
          if ( i < 5 ) printf("   %6.4lf",omega);
          }
       else
          if ( i < 5 ) printf("         ");
       }

   printf("\n");

   fclose(outfil);

   printf("   Results written to file %s\n",argv[3]);
   printf("   CPU memory allocated  = %.2lf MB\n",CPUMemory/1000000.0);
   printf("   GPU memory allocated  = %.2lf MB\n",GPUMemory/1000000.0);

   gettimeofday(&_ttime, &_tzone);
   walltime = (double)(_ttime.tv_sec) + (double)(_ttime.tv_usec/1000000.0) - walltime;

   printf("   Total wall clock time = %.2lf s\n", walltime);

    
   // free host and device memory
   free(real_rasc); free(real_decl);
   free(rand_rasc); free(rand_decl);
   hipFree(d_real_rasc); hipFree(d_real_decl);
   hipFree(d_rand_rasc); hipFree(d_rand_decl);

   free(histogramDR); free(histogramDD); free(histogramRR);
   hipFree(d_histogramDR); hipFree(d_histogramDD); hipFree(d_histogramRR);
    
   return(0);
}

int parseargs_readinput(int argc, char *argv[])
    {
    FILE *real_data_file, *rand_data_file, *out_file;
    float arcmin2rad = 1.0f/60.0f/180.0f*pif;
    int Number_of_Galaxies;
  
    if ( argc != 4 )
       {
       printf("   Usage: galaxy real_data random_data output_file\n   All MPI processes will be killed\n");
       return(1);
       }
    if ( argc == 4 )
       {
       printf("   Running galaxy_openmp %s %s %s\n",argv[1], argv[2], argv[3]);

       real_data_file = fopen(argv[1],"r");
       if ( real_data_file == NULL )
          {
          printf("   Usage: galaxy  real_data  random_data  output_file\n");
          printf("   ERROR: Cannot open real data file %s\n",argv[1]);
          return(1);
          }
       else
      {
          fscanf(real_data_file,"%d",&Number_of_Galaxies);
          if ( Number_of_Galaxies != 100000L )
             {
             printf("Cannot read file %s correctly, first item not 100000\n",argv[1]);
             fclose(real_data_file);
             return(1);
             }
          for ( int i = 0; i < NUM_OF_GALAXIES; ++i )
              {
                float rasc, decl;
          if ( fscanf(real_data_file,"%f %f", &rasc, &decl ) != 2 )
             {
                 printf("   ERROR: Cannot read line %d in real data file %s\n",i+1,argv[1]);
                 fclose(real_data_file);
             return(1);
             }
          real_rasc[i] = rasc*arcmin2rad;
          real_decl[i] = decl*arcmin2rad;
          }
           fclose(real_data_file);
       printf("   Successfully read 100000 lines from %s\n",argv[1]);
       }

       rand_data_file = fopen(argv[2],"r");
       if ( rand_data_file == NULL )
          {
          printf("   Usage: galaxy  real_data  random_data  output_file\n");
          printf("   ERROR: Cannot open random data file %s\n",argv[2]);
          return(1);
          }
       else
      {
          fscanf(rand_data_file,"%d",&Number_of_Galaxies);
          if ( Number_of_Galaxies != 100000L )
             {
             printf("Cannot read file %s correctly, first item not 100000\n",argv[2]);
             fclose(rand_data_file);
             return(1);
             }
          for ( int i = 0; i < NUM_OF_GALAXIES; ++i )
              {
                float rasc, decl;
          if ( fscanf(rand_data_file,"%f %f", &rasc, &decl ) != 2 )
             {
                 printf("   ERROR: Cannot read line %d in real data file %s\n",i+1,argv[2]);
                 fclose(rand_data_file);
             return(1);
             }
          rand_rasc[i] = rasc*arcmin2rad;
          rand_decl[i] = decl*arcmin2rad;
          }
          fclose(rand_data_file);
      printf("   Successfully read 100000 lines from %s\n",argv[2]);
      }
       out_file = fopen(argv[3],"w");
       if ( out_file == NULL )
          {
          printf("   Usage: galaxy  real_data  random_data  output_file\n");
          printf("   ERROR: Cannot open output file %s\n",argv[3]);
          return(1);
          }
       else fclose(out_file);
       }

    return(0);
    }




int getDevice(void)
{

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("   Found %d CUDA devices\n",deviceCount);
  if ( deviceCount < 0 || deviceCount > 128 ) return(-1);
  int device;
  for (device = 0; device < deviceCount; ++device) {
       hipDeviceProp_t deviceProp;
       hipGetDeviceProperties(&deviceProp, device);
       printf("      Device %s                  device %d\n", deviceProp.name,device);
       printf("         compute capability           =         %d.%d\n", deviceProp.major, deviceProp.minor);
       printf("         totalGlobalMemory            =        %.2lf GB\n", deviceProp.totalGlobalMem/1000000000.0);
       printf("         l2CacheSize                  =    %8d B\n", deviceProp.l2CacheSize);
       printf("         regsPerBlock                 =    %8d\n", deviceProp.regsPerBlock);
       printf("         multiProcessorCount          =    %8d\n", deviceProp.multiProcessorCount);
       printf("         maxThreadsPerMultiprocessor  =    %8d\n", deviceProp.maxThreadsPerMultiProcessor);
       printf("         sharedMemPerBlock            =    %8d B\n", (int)deviceProp.sharedMemPerBlock);
       printf("         warpSize                     =    %8d\n", deviceProp.warpSize);
       printf("         clockRate                    =    %8.2lf MHz\n", deviceProp.clockRate/1000.0);
       printf("         maxThreadsPerBlock           =    %8d\n", deviceProp.maxThreadsPerBlock);
       printf("         asyncEngineCount             =    %8d\n", deviceProp.asyncEngineCount);
       printf("         f to lf performance ratio    =    %8d\n", deviceProp.singleToDoublePrecisionPerfRatio);
       printf("         maxGridSize                  =    %d x %d x %d\n",
                          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
       printf("         maxThreadsDim                =    %d x %d x %d\n",
                          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
       printf("         concurrentKernels            =    ");
       if(deviceProp.concurrentKernels==1) printf("     yes\n"); else printf("    no\n");
       printf("         deviceOverlap                =    %8d\n", deviceProp.deviceOverlap);
       if(deviceProp.deviceOverlap == 1)
       printf("            Concurrently copy memory/execute kernel\n");
       }

    hipSetDevice(0);
    hipGetDevice(&device);
    if ( device != 0 ) printf("   Unable to set device 0, using %d instead",device);
    else printf("   Using CUDA device %d\n\n", device);

return(0);
}


